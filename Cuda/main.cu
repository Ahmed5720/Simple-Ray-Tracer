#include "hip/hip_runtime.h"
int main(){
	
	int num_pixels = nx*ny;
	size_t fb_size = 3 * num_pixels * sizeof(float); //allocate rgb floats for each pixel

	float *fb; 
	checkCudaErrors(hipMallocManaged((void**)&fb), fb_size));

	int tx = 8; 
	int ty = 8; // we use blocks of 8*8 threads **multiple of 32 so can fit in warps



	dim3 blocks(nx/tx+1, ny/ty+1);
	dim3 threads(tx,ty);






}


__global__ void render(float *fb, int max_x, int max_y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(())
}

