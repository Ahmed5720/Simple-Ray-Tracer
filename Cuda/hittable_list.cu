#include "hip/hip_runtime.h"
//
// Created by 15854 on 3/14/2024.
//

#ifndef RAYTRACER_HITTABLE_LIST_H
#define RAYTRACER_HITTABLE_LIST_H
#include "hittable.h"
#include "interval.h"

#include <memory>
#include <vector>


using namespace std;

class hittable_list : public hittable {
public:
    vector<shared_ptr<hittable>> hittable_objects;

    hittable_list(){

    }
    hittable_list(shared_ptr<hittable> object)
    {
        add(object);
    }

    void add(shared_ptr<hittable> obj)
    {
        hittable_objects.push_back(obj);
    }

    bool hit(const ray& r, interval ray_t, hit_record& rec) const override {
        hit_record tmp;
        bool hit_any = false;
        auto closest_hit = ray_t.max;

        for (const auto& object : hittable_objects)
        {
            if(object->hit(r, interval(ray_t.min, closest_hit), tmp))
            {
                hit_any = true;
                closest_hit = tmp.t;
                rec = tmp;
            }
        }

        return hit_any;



    }

};
#endif //RAYTRACER_HITTABLE_LIST_H
